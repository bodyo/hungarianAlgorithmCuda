
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <iostream>
using namespace std;

typedef pair<int, int> PInt;
typedef vector<int> VInt;
typedef vector<VInt> VVInt;
typedef vector<PInt> VPInt;

const int inf = numeric_limits<int>::max();

/*
 * Решает задачу о назначениях Венгерским методом.
 * matrix: прямоугольная матрица из целых чисел (не обязательно положительных).
 *         Высота матрицы должна быть не больше ширины.
 * Возвращает: Список выбранных элементов, по одному из каждой строки матрицы.
 */
VPInt hungarian(const VVInt &matrix) {
   
   // Размеры матрицы
   int height = matrix.size(), width = matrix[0].size();
   
   // Значения, вычитаемые из строк (u) и столбцов (v)
   VInt u(height, 0), v(width, 0);
   
   // Индекс помеченной клетки в каждом столбце
   VInt markIndices(width, -1);
   
   // Будем добавлять строки матрицы одну за другой
   for(int i = 0; i < height; i++) {
      VInt links(width, -1);
      VInt mins(width, inf);
      VInt visited(width, 0);
      
      // Разрешение коллизий (создание "чередующейся цепочки" из нулевых элементов)
      int markedI = i, markedJ = -1, j;
      while(markedI != -1) {
         // Обновим информацию о минимумах в посещенных строках непосещенных столбцов
         // Заодно поместим в j индекс непосещенного столбца с самым маленьким из них
         j = -1;
         for(int j1 = 0; j1 < width; j1++)
            if(!visited[j1]) {
               if(matrix[markedI][j1] - u[markedI] - v[j1] < mins[j1]) {
                  mins[j1] = matrix[markedI][j1] - u[markedI] - v[j1];
                  links[j1] = markedJ;
               }
               if(j==-1 || mins[j1] < mins[j])
                  j = j1;
            }
            
         // Теперь нас интересует элемент с индексами (markIndices[links[j]], j)
         // Произведем манипуляции со строками и столбцами так, чтобы он обнулился
         int delta = mins[j];
         for(int j1 = 0; j1 < width; j1++)
            if(visited[j1]) {
               u[markIndices[j1]] += delta;
               v[j1] -= delta;
            } else {
               mins[j1] -= delta;
            }
         u[i] += delta;
         
         // Если коллизия не разрешена - перейдем к следующей итерации
         visited[j] = 1;
         markedJ = j;
         markedI = markIndices[j];   
      }
      
      // Пройдем по найденной чередующейся цепочке клеток, снимем отметки с
      // отмеченных клеток и поставим отметки на неотмеченные
      for(; links[j] != -1; j = links[j])
         markIndices[j] = markIndices[links[j]];
      markIndices[j] = i;
   }
   
   // Вернем результат в естественной форме
   VPInt result;
   for(int j = 0; j < width; j++)
      if(markIndices[j] != -1)
         result.push_back(PInt(markIndices[j], j));
   return result;
}

int main(){
    
    int lengthOfColumn;
    int lengthOfRow;
    do
    {
    	cout << "Enter length of column" << endl;
    	cin >> lengthOfColumn;
    	cout << "Enter length of row" << endl;
    	cin >> lengthOfRow;
    }
    while (lengthOfColumn > lengthOfRow);

    vector<vector<int> > matrixOfEmployeeWork;

    matrixOfEmployeeWork.reserve(lengthOfColumn);
    for(int i = 0; i < lengthOfRow; ++i)
   		matrixOfEmployeeWork[i].reserve(lengthOfRow);

    for(int i = 0; i < lengthOfColumn; ++i)
    	for(int j = 0; j < lengthOfRow; ++j)
    		cin >> matrixOfEmployeeWork[i][j];

  //  	int m[][11] = {
		// {INF, 	7858, 8743, 17325, 18510, 9231,  4920, 7056, 9701, 5034, 7825}, 
  //       {8128, 	INF,  5021, 13603, 19635, 11386, 7075, 8840, 1843, 7189, 9256}, 
  //       {6809, 	5364, INF,  8582,  14614, 10067, 5756, 5904, 7207, 3882, 4235}, 
  //       {7849, 	5515, 1040, INF,   15654, 11107, 6796, 4713, 7358, 4900, 5275}, 
  //       {10918, 8365, 4109, 5808,  INF,   14176, 9865, 7928, 931,  7991, 8344}, 
  //       {336, 	7285, 2830, 11412, 17444, INF,   4347, 6483, 6688, 4461, 7065}, 
  //       {1053, 	2938, 3823, 12405, 15835, 4311,  INF,  2136, 4781, 114,  2905}, 
  //       {8930, 	802,  5823, 14405, 20437, 12188, 7877, INF,  2645, 7429, 10058}, 
  //       {9987, 	7434, 3178, 11760, 17792, 13245, 8934, 6997, INF,  7060, 7413}, 
  //       {10518, 2824, 3709, 12291, 15721, 13776, 9465, 2022, 4667, INF,  7944}, 
  //       {2574, 	4459, 5344, 9561,  17356, 5832,  1521, 3657, 6302, 1635, INF}
  //  };
   VVInt matrix;
   for(int i=0; i<lengthOfColumn; i++)
   {
    	VInt row;
     	for(int j=0; j<lengthOfRow; j++) 
     		row.push_back(matrixOfEmployeeWork[i][j]);  
     	matrix.push_back(row);
   }
   cout << "debug" << endl;
   VPInt result;
   long t1 = clock();
   for(int i=0; i < 10000; i++)
   	result = hungarian(matrix);

   t1 = clock() - t1;

   int cost = 0;
   for(int i = 0; i < result.size(); i++)
   {
    	pair <int,int> e = result[i];
        cout << e.first << "->" << e.second << endl;
        cost += matrixOfEmployeeWork[e.first][e.second];
   }
   cout << "Cost     : " << cost << endl;
   cout << "Time (ms): " <<  t1 << endl;

   return 0;
}