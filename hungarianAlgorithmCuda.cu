
#include <hip/hip_runtime.h>
#include <vector>
#include <limits>
#include <iostream>
using namespace std;

typedef pair<int, int> PInt;
typedef vector<int> VInt;
typedef vector<VInt> VVInt;
typedef vector<PInt> VPInt;

const int inf = numeric_limits<int>::max();

VPInt hungarian(const VVInt &matrix) {
   
   
   int height = matrix.size(), width = matrix[0].size();
   
   
   VInt u(height, 0), v(width, 0);
   
   
   VInt markIndices(width, -1);
   
   
   for(int i = 0; i < height; i++) {
      VInt links(width, -1);
      VInt mins(width, inf);
      VInt visited(width, 0);
      
      
      int markedI = i, markedJ = -1, j;
      while(markedI != -1) {
         
         j = -1;
         for(int j1 = 0; j1 < width; j1++)
            if(!visited[j1]) {
               if(matrix[markedI][j1] - u[markedI] - v[j1] < mins[j1]) {
                  mins[j1] = matrix[markedI][j1] - u[markedI] - v[j1];
                  links[j1] = markedJ;
               }
               if(j==-1 || mins[j1] < mins[j])
                  j = j1;
            }
            
         int delta = mins[j];
         for(int j1 = 0; j1 < width; j1++)
            if(visited[j1]) {
               u[markIndices[j1]] += delta;
               v[j1] -= delta;
            } else {
               mins[j1] -= delta;
            }
         u[i] += delta;
         
         
         visited[j] = 1;
         markedJ = j;
         markedI = markIndices[j];   
      }
      
      
      for(; links[j] != -1; j = links[j])
         markIndices[j] = markIndices[links[j]];
      markIndices[j] = i;
   }
   
   VPInt result;
   for(int j = 0; j < width; j++)
      if(markIndices[j] != -1)
         result.push_back(PInt(markIndices[j], j));
   return result;
}

int main(){
    
    int lengthOfColumn;
    int lengthOfRow;
    do
    {
    	cout << "Enter length of column" << endl;
    	cin >> lengthOfColumn;
    	cout << "Enter length of row" << endl;
    	cin >> lengthOfRow;
    }
    while (lengthOfColumn > lengthOfRow);
    cout << "Input elements of matrix" << endl;

    vector<vector<int> > matrixOfEmployeeWork;

    matrixOfEmployeeWork.reserve(lengthOfColumn);
    for(int i = 0; i < lengthOfRow; ++i)
   		matrixOfEmployeeWork[i].reserve(lengthOfRow);

    for(int i = 0; i < lengthOfColumn; ++i)
    	for(int j = 0; j < lengthOfRow; ++j)
    		cin >> matrixOfEmployeeWork[i][j];

   VVInt matrix;
   for(int i=0; i<lengthOfColumn; i++)
   {
    	VInt row;
     	for(int j=0; j<lengthOfRow; j++) 
     		row.push_back(matrixOfEmployeeWork[i][j]);  
     	matrix.push_back(row);
   }
   cout << "debug" << endl;
   VPInt result;
   long t1 = clock();
   for(int i=0; i < 10000; i++)
   	result = hungarian(matrix);

   t1 = clock() - t1;

   int cost = 0;
   for(int i = 0; i < result.size(); i++)
   {
    	pair <int,int> e = result[i];
        cout << e.first << "->" << e.second << endl;
        cost += matrixOfEmployeeWork[e.first][e.second];
   }
   cout << "Cost     : " << cost << endl;
   cout << "Time (ms): " <<  t1 << endl;

   return 0;
}