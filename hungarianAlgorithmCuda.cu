#include "hip/hip_runtime.h"
// #include <vector>
// #include <limits>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/pair.h>
#include <thrust/device_reference.h>
#include <thrust/memory.h>
// #include <thrust/


using namespace std;
using namespace thrust;

typedef thrust::pair<int, int> PInt;
typedef thrust::device_vector<int> VInt;
typedef thrust::device_vector<VInt> VVInt;
typedef thrust::device_vector<PInt> VPInt;

const int inf = numeric_limits<int>::max();

__global__
void hungarian(VVInt *matrix, VPInt *result) 
{
   int height = matrix->size(), width = (*matrix)[0]->size();
   VInt u(height, 0), v(width, 0);
   VInt markIndices(width, -1);
   
   for(int i = 0; i < height; i++) {
      VInt links(width, -1);
      VInt mins(width, inf);
      VInt visited(width, 0);
      
      int markedI = i, markedJ = -1, j;
      while(markedI != -1) {
         
         j = -1;
         for(int j1 = 0; j1 < width; j1++)
            if(!visited[j1]) 
            {
               if((*matrix)[markedI][j1] - u[markedI] - v[j1] < mins[j1]) 
               {
                  mins[j1] = (*matrix)[markedI][j1] - u[markedI] - v[j1];
                  links[j1] = markedJ;
               }
               if(j==-1 || mins[j1] < mins[j])
                  j = j1;
            }
            
         int delta = mins[j];
         for(int j1 = 0; j1 < width; j1++)
            if(visited[j1]) {
               u[markIndices[j1]] += delta;
               v[j1] -= delta;
            } else {
               mins[j1] -= delta;
            }
         u[i] += delta;
         
         visited[j] = 1;
         markedJ = j;
         markedI = markIndices[j];   
      }
      
      for(; links[j] != -1; j = links[j])
         markIndices[j] = markIndices[links[j]];
      markIndices[j] = i;
   }

   for(int j = 0; j < width; j++)
      if(markIndices[j] != -1)
         result->push_back(PInt(markIndices[j], j));
}

int main(){
    
    int lengthOfColumn;
    int lengthOfRow;
    do
    {
    	cout << "Enter length of column" << endl;
    	cin >> lengthOfColumn;
    	cout << "Enter length of row" << endl;
    	cin >> lengthOfRow;
    }
    while (lengthOfColumn > lengthOfRow);
    cout << "Input elements of matrix" << endl;

    vector<vector<int> > matrixOfEmployeeWork;

    matrixOfEmployeeWork.reserve(lengthOfColumn);
    for(int i = 0; i < lengthOfRow; ++i)
   		matrixOfEmployeeWork[i].reserve(lengthOfRow);

    for(int i = 0; i < lengthOfColumn; ++i)
    	for(int j = 0; j < lengthOfRow; ++j)
    		cin >> matrixOfEmployeeWork[i][j];

   	VVInt *matrix = new VVInt;
   	for(int i=0; i<lengthOfColumn; i++)
   	{
   	 	VInt row;
   	  	for(int j=0; j<lengthOfRow; j++) 
   	  		row.push_back(matrixOfEmployeeWork[i][j]);  
   	  	matrix->push_back(row);
   	}

   	VPInt *result = new VPInt;
   	long t1 = clock();

   	// Run kernel on 1M elements on the GPU
  	hungarian<<<1, 1>>>(matrix, result);

  	// Wait for GPU to finish before accessing on host
  	hipDeviceSynchronize();
		
   	t1 = clock() - t1;
		
   	int cost = 0;
   	for(int i = 0; i < result->size(); i++)
   	{
    	PInt e = (*result)[i];
   	    cout << e.first << "->" << e.second << endl;
        cost += matrixOfEmployeeWork[e.first][e.second];
   	}
   	cout << "Cost     : " << cost << endl;
   	cout << "Time (ms): " <<  t1 << endl;

   return 0;
}