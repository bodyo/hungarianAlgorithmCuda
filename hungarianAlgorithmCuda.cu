/* http://acm.mipt.ru/twiki/bin/view/Algorithms/HungarianAlgorithmCPP 
 * Âåíãåðñêèé àëãîðèòì.
 * Äàíèèë Øâåä, 2008. danshved [no-spam] gmail.com
 * Ðåàëèçàöèÿ íàâåÿíà ïñåâäîêîäîì À.Ñ.Ëîïàòèíà èç êíèãè
 * "Îïòèìèçàöèÿ íà ãðàôàõ (àëãîðèòìû è ðåàëèçàöèÿ)".
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <iterator>
#include <vector>
#include <limits>
#include <time.h>

using namespace std;

typedef pair<int, int> PInt;
typedef vector<int> VInt;
typedef vector<VInt> VVInt;
typedef vector<PInt> VPInt;

#define INF 1000000000

const int inf = numeric_limits<int>::max();

/*
 * Ðåøàåò çàäà÷ó î íàçíà÷åíèÿõ Âåíãåðñêèì ìåòîäîì.
 * matrix: ïðÿìîóãîëüíàÿ ìàòðèöà èç öåëûõ ÷èñåë (íå îáÿçàòåëüíî ïîëîæèòåëüíûõ).
 *         Âûñîòà ìàòðèöû äîëæíà áûòü íå áîëüøå øèðèíû.
 * Âîçâðàùàåò: Ñïèñîê âûáðàííûõ ýëåìåíòîâ, ïî îäíîìó èç êàæäîé ñòðîêè ìàòðèöû.
 */
VPInt hungarian(const VVInt &matrix) {
   
   // Ðàçìåðû ìàòðèöû
   int height = matrix.size(), width = matrix[0].size();
   
   // Çíà÷åíèÿ, âû÷èòàåìûå èç ñòðîê (u) è ñòîëáöîâ (v)
   VInt u(height, 0), v(width, 0);
   
   // Èíäåêñ ïîìå÷åííîé êëåòêè â êàæäîì ñòîëáöå
   VInt markIndices(width, -1);
   
   // Áóäåì äîáàâëÿòü ñòðîêè ìàòðèöû îäíó çà äðóãîé
   for(int i = 0; i < height; i++) {

      VInt links(width, -1);
      VInt mins(width, inf);
      VInt visited(width, 0);
      
      // Ðàçðåøåíèå êîëëèçèé (ñîçäàíèå "÷åðåäóþùåéñÿ öåïî÷êè" èç íóëåâûõ ýëåìåíòîâ)
      int markedI = i, markedJ = -1, j;
      do{
         // Îáíîâèì èíôîðìàöèþ î ìèíèìóìàõ â ïîñåùåííûõ ñòðîêàõ íåïîñåùåííûõ ñòîëáöîâ
         // Çàîäíî ïîìåñòèì â j èíäåêñ íåïîñåùåííîãî ñòîëáöà ñ ñàìûì ìàëåíüêèì èç íèõ
         j = -1;
         for(int j1 = 0; j1 < width; j1++)
            if(!visited[j1]) {
               if(matrix[markedI][j1] - u[markedI] - v[j1] < mins[j1]) {
                  mins[j1] = matrix[markedI][j1] - u[markedI] - v[j1];
                  links[j1] = markedJ;
               }
               if(j==-1 || mins[j1] < mins[j])
                  j = j1;
            }
            
         // Òåïåðü íàñ èíòåðåñóåò ýëåìåíò ñ èíäåêñàìè (markIndices[links[j]], j)
         // Ïðîèçâåäåì ìàíèïóëÿöèè ñî ñòðîêàìè è ñòîëáöàìè òàê, ÷òîáû îí îáíóëèëñÿ
         int delta = mins[j];
         for(int j1 = 0; j1 < width; j1++)
            if(visited[j1]) {
               u[markIndices[j1]] += delta;
               v[j1] -= delta;
            } else {
               mins[j1] -= delta;
            }
         u[i] += delta;
         
         // Åñëè êîëëèçèÿ íå ðàçðåøåíà - ïåðåéäåì ê ñëåäóþùåé èòåðàöèè
         visited[j] = 1;
         markedJ = j;
         markedI = markIndices[j];   

      }while(markedI != -1);

      // Ïðîéäåì ïî íàéäåííîé ÷åðåäóþùåéñÿ öåïî÷êå êëåòîê, ñíèìåì îòìåòêè ñ
      // îòìå÷åííûõ êëåòîê è ïîñòàâèì îòìåòêè íà íåîòìå÷åííûå
      for(; links[j] != -1; j = links[j])
         markIndices[j] = markIndices[links[j]];
      markIndices[j] = i;
   }
   
   // Âåðíåì ðåçóëüòàò â åñòåñòâåííîé ôîðìå
   VPInt result;
   for(int j = 0; j < width; j++)
      if(markIndices[j] != -1)
         result.push_back(PInt(markIndices[j], j));
   return result;
}

int main(){
    
   int m[][11] = {
	{INF, 7858, 8743, 17325, 18510, 9231, 4920, 7056, 9701, 5034, 7825}, 
        {8128, INF, 5021, 13603, 19635, 11386, 7075, 8840, 1843, 7189, 9256}, 
        {6809, 5364, INF, 8582, 14614, 10067, 5756, 5904, 7207, 3882, 4235}, 
        {7849, 5515, 1040, INF, 15654, 11107, 6796, 4713, 7358, 4900, 5275}, 
        {10918, 8365, 4109, 5808, INF, 14176, 9865, 7928, 931, 7991, 8344}, 
        {336, 7285, 2830, 11412, 17444, INF, 4347, 6483, 6688, 4461, 7065}, 
        {1053, 2938, 3823, 12405, 15835, 4311, INF, 2136, 4781, 114, 2905}, 
        {8930, 802, 5823, 14405, 20437, 12188, 7877, INF, 2645, 7429, 10058}, 
        {9987, 7434, 3178, 11760, 17792, 13245, 8934, 6997, INF, 7060, 7413}, 
        {10518, 2824, 3709, 12291, 15721, 13776, 9465, 2022, 4667, INF, 7944}, 
        {2574, 4459, 5344, 9561, 17356, 5832, 1521, 3657, 6302, 1635, INF}
   };
   VVInt matrix;
   for(int i=0; i<11; i++)
   {
    	VInt row;
     	for(int j=0; j<11; j++) row.push_back(m[i][j]);  
     	matrix.push_back(row);
   }

   VPInt result;
   long t1 = clock();
   for(int i=0; i < 10000; i++)
   	result = hungarian(matrix);

   t1 = clock() - t1;

   int cost = 0;
   for(int i = 0; i < result.size(); i++)
   {
    	pair <int,int> e = result[i];
        cout << e.first << "->" << e.second << endl;
        cost += m[e.first][e.second];
   }
   cout << "Cost     : " << cost << endl;
   cout << "Time (ms): " <<  t1 << endl;

   return 0;
}